#include "hip/hip_runtime.h"
//
// Created by JEONGHYUNLEE on 2020/09/27.
//

#include "caffe/caffe.hpp"
#include "common.hpp"
#include <stdio.h>

using namespace caffe;
typedef float Dtype;

// Kernel Function
__global__ void replace_kernel(const int n,
                               Dtype *target,
                               const Dtype *idx_mask,
                               const Dtype value) {
  CUDA_KERNEL_LOOP(index, n)
  {
    if (idx_mask[index] == 1.0) {
      target[index] = value;
    }
  }
}

// Wrapper
void caffe_gpu_replace(const int N,
                       Dtype *target,
                       const Dtype *idx_mask,
                       const Dtype value) {
  replace_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, target, idx_mask, value);
}

// Test Code
int main() {
  vector<int> weight_shape = {2, 2, 2, 2};
  shared_ptr <Blob<Dtype>> blob_a(new Blob<Dtype>(weight_shape));
  shared_ptr <Blob<Dtype>> blob_b(new Blob<Dtype>(weight_shape));

  set_values(blob_a->mutable_cpu_data(), {0, 0, 0, 0,
                                          1, 1, 1, 1,
                                          2, 2, 2, 2,
                                          3, 3, 3, 3}, blob_a->count());

  set_values(blob_b->mutable_cpu_data(), {1, 1, 0, 0,
                                          0, 1, 0, 1,
                                          1, 0, 1, 0,
                                          0, 0, 1, 1}, blob_b->count());

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());
  print_tensor("b", blob_b->cpu_data(), blob_b->shape());

  CHECK_EQ(blob_a->count(), blob_b->count());
  caffe_gpu_replace(blob_a->count(), blob_a->mutable_gpu_data(), blob_b->mutable_gpu_data(), 100.0);

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());
  print_tensor("b", blob_b->cpu_data(), blob_b->shape());

  return 0;
}

