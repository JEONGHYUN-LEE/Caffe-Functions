#include "hip/hip_runtime.h"
//
// Created by JEONGHYUNLEE on 2020/09/28.
//

#include <stdio.h>
#include <caffe/caffe.hpp>
#include "./common.hpp"

using namespace caffe;
typedef float Dtype;


// Wrapper
Dtype caffe_gpu_filter_norm(const std::vector<int> shape,
                            const Dtype *target,
                            const int filter_idx) {
  const int filter_size = shape[1] * shape[2] * shape[3];
  Dtype result;
  CUBLAS_CHECK(hipblasSnrm2(Caffe::cublas_handle(),
                           filter_size,
                           target + filter_idx * filter_size,
                           1,
                           &result));
  return result;
}


// Test Code
int main() {
  vector<int> weight_shape = {2, 2, 2, 2};
  shared_ptr <Blob<Dtype>> blob_a(new Blob<Dtype>(weight_shape));

  set_values(blob_a->mutable_cpu_data(), {0, 0, 0, 0,
                                          1, 1, 1, 1,
                                          2, 2, 2, 2,
                                          3, 3, 3, 3}, blob_a->count());

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());

  Dtype result = 0.0;

  //it should be smaller than the number of filters (2)
  const int filter_index = 1;

  result = caffe_gpu_filter_norm(blob_a->shape(), blob_a->gpu_data(), filter_index);

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());

  printf("%dth filter norm: %f", filter_index, result);
  printf("\n");

  return 0;
}
