#include "hip/hip_runtime.h"
//
// Created by JEONGHYUNLEE on 2020/09/28.
//

#include <stdio.h>
#include <caffe/caffe.hpp>
#include "./common.hpp"

using namespace caffe;
typedef float Dtype;


// Wrapper
Dtype caffe_gpu_filter_norm(const int N,
                            const Dtype *target,
                            const int filter_size,
                            const int filter_idx) {
  Dtype result = 0.0;
  CUBLAS_CHECK(hipblasSnrm2(Caffe::cublas_handle(), filter_size, target + filter_idx * filter_size, 1, &result));
  return result;
}


// Test Code
int main() {
  vector<int> weight_shape = {2,2,2,2};
  shared_ptr<Blob<Dtype> > blob_a(new Blob<Dtype>(weight_shape));

  set_values(blob_a->mutable_cpu_data(), {0,0,0,0,
                                          1,1,1,1,
                                          2,2,2,2,
                                          3,3,3,3}, blob_a->count());

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());

  Dtype result = 0.0;

  const int filter_size = blob_a->channels()*blob_a->height()*blob_a->width();

  //it should be smaller than the number of filters (2)
  const int filter_index = 1;

  result = caffe::caffe_gpu_filter_norm(blob_a->count(), blob_a->gpu_data(), filter_size, filter_index);

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());

  printf("%dth filter norm: %f", filter_index, result);
  printf("\n");

  return 0;
}
