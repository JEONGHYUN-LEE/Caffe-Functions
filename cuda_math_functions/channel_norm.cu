#include "hip/hip_runtime.h"
//
// Created by JEONGHYUNLEE on 2020/09/28.
//

#include <stdio.h>
#include <caffe/caffe.hpp>
#include "./common.hpp"

using namespace caffe;
typedef float Dtype;


// Wrapper
Dtype caffe_gpu_channel_norm(const std::vector<int> shape,
                             const Dtype *target,
                             const int channel_idx) {
  const int filter_number = shape[0];
  const int filter_size = shape[1] * shape[2] * shape[3];
  const int channel_size = shape[2] * shape[3];
  Dtype filter_result;
  Dtype result = 0;
  for (int filter_idx = 0; filter_idx < filter_number; filter_idx++) {
    CUBLAS_CHECK(hipblasSnrm2(Caffe::cublas_handle(),
                             channel_size,
                             target + filter_idx * filter_size+channel_idx*channel_size,
                             1,
                             &filter_result));
    result+=filter_result*filter_result;
  }
  result = sqrt(result);
  return result;
}


// Test Code
int main() {
  vector<int> weight_shape = {2, 2, 2, 2};
  shared_ptr <Blob<Dtype>> blob_a(new Blob<Dtype>(weight_shape));

  set_values(blob_a->mutable_cpu_data(), {0, 0, 0, 0,
                                          1, 1, 1, 1,
                                          2, 2, 2, 2,
                                          3, 3, 3, 3}, blob_a->count());

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());

  Dtype result = 0.0;

  //it should be smaller than the number of channels (2)
  const int filter_index = 1;

  result = caffe_gpu_channel_norm(blob_a->shape(), blob_a->gpu_data(), filter_index);

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());

  printf("%dth channel norm: %f", filter_index, result);
  printf("\n");

  return 0;
}
