#include "hip/hip_runtime.h"
//
// Created by JEONGHYUNLEE on 2020/09/27.
//

#include "caffe/caffe.hpp"
#include "./common.hpp"
#include <stdio.h>

using namespace caffe;
typedef float Dtype;

// Kernel Function
__global__ void masking_kernel(const int n,
                               Dtype *a,
                               const Dtype *b
) {
  CUDA_KERNEL_LOOP(index, n)
  {
    if (b[index] == 0.0) {
      a[index] = 0.0;
    }
  }
}

// Wrapper
void caffe_gpu_masking(const int N,
                       Dtype *a,
                       const Dtype *b) {
  masking_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, a, b);
}

// Test Code
int main() {
  vector<int> weight_shape = {2, 2, 2, 2};
  shared_ptr <Blob<Dtype>> blob_a(new Blob<Dtype>(weight_shape));
  shared_ptr <Blob<Dtype>> blob_b(new Blob<Dtype>(weight_shape));

  set_values(blob_a->mutable_cpu_data(), {0, 0, 0, 0,
                                          1, 1, 1, 1,
                                          2, 2, 2, 2,
                                          3, 3, 3, 3}, blob_a->count());

  set_values(blob_b->mutable_cpu_data(), {1, 1, 0, 0,
                                          0, 1, 0, 1,
                                          1, 0, 1, 0,
                                          0, 0, 1, 1}, blob_b->count());

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());
  print_tensor("b", blob_b->cpu_data(), blob_b->shape());

  CHECK_EQ(blob_a->count(), blob_b->count());
  caffe_gpu_masking(blob_a->count(), blob_a->mutable_gpu_data(), blob_b->gpu_data());

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());
  print_tensor("b", blob_b->cpu_data(), blob_b->shape());

  return 0;
}

