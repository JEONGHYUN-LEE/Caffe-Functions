#include "hip/hip_runtime.h"
//
// Created by JEONGHYUNLEE on 2020/09/28.
//

#include <stdio.h>
#include <caffe/caffe.hpp>
#include "./common.hpp"

using namespace caffe;
typedef float Dtype;

// Kernel Function
__global__ void find_kernel(const int n,
                            const Dtype* target,
                            Dtype* idx,
                            const Dtype value
){
  CUDA_KERNEL_LOOP(index, n) {
    if (target[index] == value) {
      idx[index] = 1.0;
    }
    else {
      idx[index] = 0.0;
    }
  }
}

// Wrapper
void caffe_gpu_find(const int N,
                    const Dtype* target,
                    Dtype* idx,
                    const Dtype value) {
  find_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, target, idx, value);
}

// Test Code
int main() {
  vector<int> weight_shape = {2,2,2,2};
  shared_ptr<Blob<Dtype> > blob_a(new Blob<Dtype>(weight_shape));
  shared_ptr<Blob<Dtype> > blob_b(new Blob<Dtype>(weight_shape));

  set_values(blob_a->mutable_cpu_data(), {0,0,0,0,
                                          1,1,1,1,
                                          2,2,2,2,
                                          3,3,3,3}, blob_a->count());

  set_values(blob_b->mutable_cpu_data(), {0,0,0,0,
                                          0,0,0,0,
                                          0,0,0,0,
                                          0,0,0,0}, blob_b->count());

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());
  print_tensor("b", blob_b->cpu_data(), blob_b->shape());

  CHECK_EQ(blob_a->count(), blob_b->count());

  caffe_gpu_find(blob_a->count(), blob_a->gpu_data(), blob_b->mutable_gpu_data(), 2);

  print_tensor("a", blob_a->cpu_data(), blob_a->shape());
  print_tensor("b", blob_b->cpu_data(), blob_b->shape());

  return 0;
}
